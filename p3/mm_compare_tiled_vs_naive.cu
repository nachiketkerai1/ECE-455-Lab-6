
#include <hip/hip_runtime.h>
template <typename T>
__global__ void mm_naive(const T* A, const T* B, T* C, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elems = N * N;
    if (tid >= total_elems) return;
    
    int row = tid / N;
    int col = tid % N;
    
    T val = 0;
    for (int k = 0; k < N; ++k)
        val += A[row * N + k] * B[k * N + col];
    
    C[tid] = val;
}
